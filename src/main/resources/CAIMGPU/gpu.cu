#include "hip/hip_runtime.h"
#include "parameters.h"

__global__ void computeCAIMValues(float* caimValues, int* appearance, int* intervals, int numberIntervals, int numberClasses, int numbermidPoints);

JNIEXPORT void JNICALL
Java_weka_filters_supervised_attribute_CAIMGPU_initializeGPU(JNIEnv *env, jobject obj, jobject algorithm, jint attribute, jint numberClasses, jint numberAttributes, jint numberInstances)
{
	int deviceCount, numberValues, numberIntervals, numbermidPoints, iteration = 1;
	int *h_appearance, *d_appearance, *h_classValues, *d_classValues;
	float *h_attributeValues, *d_attributeValues, *h_midpoints;
	int *h_intervals, *d_intervals, *tempInterval, *aux;
	float *h_caimValues, *d_caimValues;
	float globalCAIM = 0.0f;
	
	// Set the GPU device number and properties
	hipSetDeviceFlags(hipDeviceScheduleSpin);
	hipGetDeviceCount(&deviceCount);
	
	hipSetDevice(attribute % deviceCount);

	jclass cls = env->GetObjectClass(algorithm);
	jmethodID getAttributeValues = env->GetMethodID(cls, "getAttributeValues", "(I)[F");
	jmethodID getClassValues = env->GetMethodID(cls, "getClassValues", "()[I");
	jmethodID addInterval = env->GetMethodID(cls, "addInterval", "(IF)V");
	
	jfloatArray jattributeValues = (jfloatArray) env->CallObjectMethod(algorithm, getAttributeValues, attribute);
	jintArray jclassValues = (jintArray) env->CallObjectMethod(algorithm, getClassValues);
	
	h_attributeValues = (float*) env->GetFloatArrayElements(jattributeValues, 0);
	h_classValues = (int*) env->GetIntArrayElements(jclassValues, 0);
	
	h_appearance = (int*) calloc(numberClasses * numberInstances, sizeof(int)); 
	hipMalloc((void**) &d_attributeValues, numberInstances * sizeof(float));
	hipMalloc((void**) &d_classValues, numberInstances * sizeof(int));
	
	hipMemcpy(d_attributeValues, h_attributeValues, numberInstances * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy(d_classValues, h_classValues, numberInstances * sizeof(int), hipMemcpyHostToDevice );
	
	thrust::device_ptr<float> d_attributeValues_ptr = thrust::device_pointer_cast(d_attributeValues);
	thrust::device_ptr<int>   d_classValues_ptr = thrust::device_pointer_cast(d_classValues);
	
	thrust::sort_by_key(d_attributeValues_ptr, d_attributeValues_ptr + numberInstances, d_classValues_ptr);
	
	hipMemcpy(h_attributeValues, d_attributeValues, numberInstances * sizeof(float), hipMemcpyDeviceToHost );
	hipMemcpy(h_classValues, d_classValues, numberInstances * sizeof(int), hipMemcpyDeviceToHost );
	
	float currentValue = h_attributeValues[0];
	int offset = 0;
	numberValues = 1;
	
	for(int i = 0; i < numberInstances; i++)
	{
		if(currentValue != h_attributeValues[i])
		{
			offset++;
			
			currentValue = h_attributeValues[i];
			
			h_attributeValues[offset] = h_attributeValues[i];
			
			numberValues++;
		}
		
		h_appearance[offset*numberClasses + h_classValues[i]]++;
	}
	
	numberIntervals = 2;
	numbermidPoints = numberValues - 1;
	int numbermidPointsRemaining = numbermidPoints;
	
	h_intervals = (int*) calloc(numberClasses + 2, sizeof(int));
	tempInterval = (int*) calloc(numberClasses + 2, sizeof(int));
	h_midpoints = (float*) calloc(numberValues - 1, sizeof(float));
	h_caimValues = (float*) malloc(numbermidPoints * sizeof(float));
	
	hipMalloc((void**) &d_intervals, (numberClasses + 2) * sizeof(int));
	hipMalloc((void**) &d_appearance,  numberClasses * numberValues * sizeof(int));
	hipMalloc((void**) &d_caimValues, numbermidPoints * sizeof(float));
	
	hipMemcpy(d_appearance, h_appearance, numberClasses * numberValues * sizeof(int), hipMemcpyHostToDevice );
		
	h_intervals[0] = 0;
	h_intervals[1] = numberValues-1;
	
	for (int i = 0; i < numbermidPoints; i++)
		h_midpoints[i] = (h_attributeValues[i] + h_attributeValues[i+1]) / 2.0f;
	
	dim3 threadsCAIMValues(THREADS_EVAL_BLOCK, 1);
	dim3 gridCAIMValues((int) ceil(numbermidPoints / (THREADS_EVAL_BLOCK * 1.0f)), 1);
	
	while(1)
	{
		hipMemcpy(d_intervals, h_intervals, numberIntervals * sizeof(int), hipMemcpyHostToDevice );
		
		computeCAIMValues <<< gridCAIMValues, threadsCAIMValues >>> (d_caimValues, d_appearance, d_intervals, numberIntervals-1, numberClasses, numbermidPoints);
		
		hipMemcpy(h_caimValues, d_caimValues, numbermidPoints * sizeof(float), hipMemcpyDeviceToHost );
		
		int bestmidPoint = -1;
		float bestCAIM = -1;
		
		for(int i = 0; i < numbermidPoints; i++)
		{
			if(h_caimValues[i] > bestCAIM)
			{
				bestCAIM = h_caimValues[i];
				bestmidPoint = i;
			}
		}
		
		if (bestmidPoint == -1)		break;
		
		if(bestCAIM > globalCAIM || iteration < numberClasses)
		{
			globalCAIM = bestCAIM;
			
			for(int i = 0; i < numberIntervals; i++)
			{
				tempInterval[i] = h_intervals[i];
				
				if((bestmidPoint+1) <= h_intervals[i])
				{
					tempInterval[i] = (bestmidPoint+1);
					
					for(int j = i; j < numberIntervals; j++)
					{
						tempInterval[j+1] = h_intervals[j];
					}
					
					break;
				}
			}
			
		    aux = h_intervals;
		    h_intervals = tempInterval;
		    tempInterval = aux;
			
			iteration++;
			numberIntervals++;
			numbermidPointsRemaining--;
		}
		else
			break;
			
		if (numbermidPointsRemaining == 0) break;
	}
	
	env->CallVoidMethod(algorithm, addInterval, attribute, h_attributeValues[0]);
	
	for(int i = 1; i < numberIntervals - 1; i++)
		env->CallVoidMethod(algorithm, addInterval, attribute, h_midpoints[h_intervals[i]-1]);
		
	env->CallVoidMethod(algorithm, addInterval, attribute, h_attributeValues[numberValues-1]);
		
	env->ReleaseFloatArrayElements(jattributeValues, h_attributeValues, 0);
	env->ReleaseIntArrayElements(jclassValues, h_classValues, 0);
	
	hipFree(d_attributeValues);
	hipFree(d_classValues);
	hipFree(d_intervals);
	hipFree(d_appearance);
	hipFree(d_caimValues);
	
	free(h_appearance);
	free(h_caimValues);
	free(h_intervals);
	free(h_midpoints);
	free(tempInterval);
}

__device__ float calculate(int left, int right, int* appearance, int numberClasses)
{
	int columnSum [MAX_CLASSES] = {0};
		
	for(int i = left; i < right; i++)
	{
		for(int j = 0; j < numberClasses; j++)
		{
			columnSum[j] += appearance[i*numberClasses + j];
		}
	}
	
	int columnMax = 0, suma = 0;
	
	for(int j = 0; j < numberClasses; j++)
	{
		suma += columnSum[j];
		
		if(columnSum[j] > columnMax)
			columnMax = columnSum[j];
	} 
			
	float res = columnMax / (float) suma;
	res = res * columnMax;
	
	return res;
}

__global__ void computeCAIMValues(float* caimValues, int* appearance, int* intervals, int numberIntervals, int numberClasses, int numbermidPoints)
{
	int midPoint = blockDim.x * blockIdx.x + threadIdx.x;
	
	if(midPoint < numbermidPoints)
	{
		float CAIMValue = 0.0f;
		
		for(int k = 0; k < numberIntervals; k++)
		{
			int left = intervals[k];
			int right = intervals[k+1];
			
			if((midPoint+1) == left)
			{
				caimValues[midPoint] = 0.0f;
				return;
			}
			
			if(k == numberIntervals-1)	right++;
			
			if(left <= midPoint && midPoint < right)
			{
				CAIMValue += calculate(left, midPoint+1, appearance, numberClasses);
				CAIMValue += calculate(midPoint+1, right, appearance, numberClasses);
			}
			else
			{
				CAIMValue += calculate(left, right, appearance, numberClasses);
			}
		}
		
		caimValues[midPoint] = CAIMValue / (float) (numberIntervals+1);
	}
}